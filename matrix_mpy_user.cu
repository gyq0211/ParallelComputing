#include "hip/hip_runtime.h"
#include <iostream>
#include "ee155_utils.hxx"
#include "matrix.hxx"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
using namespace std;
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
const int BS = 32;	// The blocks are BS x BS.


///////////////////////////////
// This is the CUDA kernel function for you to write.
//
__global__ void mat_mult (float *d_A, float *d_B, float *d_C, int N) {
    //int rB=blockIdx...,
    int rB = blockIdx.x;
    int cB = blockIdx.y;
    int rI = threadIdx.x;
    int cI = threadIdx.y;

    __shared__ float SA[BS][BS], SB[BS][BS];
    //printf("In thread with r=(%d,%d) c=(%d,%d)\n", rB,rI,cB,cI);

    //...
    
    float Temp = 0.0;
	SA[cI][rI] = 0.0;
	SB[cI][rI] = 0.0;
    for (size_t kB=0; kB<N; ++kB)
    {
        SA[cI][rI] = *(d_A + rB * BS * N * BS + cI * N * BS + kB * BS + rI);
        SB[cI][rI] = *(d_B + kB * BS * N * BS + cI * N * BS + cB * BS + rI);
		__syncthreads();
        for (size_t j=0; j<BS; ++j)
        {
            Temp += ((int)(SA[cI][j]))*SB[j][rI];
            __syncthreads();
        }
    }
    *(d_C + rB * BS * N * BS + cI * N * BS + cB * BS + rI) = Temp;
}


///////////////////////////////
// This is the host function for you to write.
// It allocates memory and moves data between CPU<->GPU
void Matrix::mpy1 (const Matrix &A, const Matrix &B, int BS) {

    // Copy A from host memory to device memory.
    int numElem=N()*N(), sizeBytes = numElem*4;
    float *d_A = NULL;
    hipError_t err = hipMalloc((void **)&d_A, sizeBytes);
    ERR_CHK (err, "Failed to allocate device matrix A");

    err = hipMemcpy (d_A, A.data.data(), sizeBytes, hipMemcpyHostToDevice);
    ERR_CHK (err, "Failed to copy matrix A from host to device");

    // Allocate device memory for B.
    //...
    float* d_B = NULL;
    err = hipMalloc((void**)&d_B, sizeBytes);
    ERR_CHK(err, "Failed to allocate device matrix B");

    // Copy B from host memory to device memory.
    //...
    err = hipMemcpy(d_B, B.data.data(), sizeBytes, hipMemcpyHostToDevice);
    ERR_CHK(err, "Failed to copy matrix B from host to device");

    // Allocate device memory for C.
    //...
    float* d_C = NULL;
    err = hipMalloc((void**)&d_C, sizeBytes);
    ERR_CHK(err, "Failed to allocate device matrix C");

    // Launch the CUDA Kernel
    //...
    int NBLK = this->N() / BS;
    dim3 thBlocks(NBLK, NBLK), threads(BS, BS);
    mat_mult <<<thBlocks, threads>>> (d_A, d_B, d_C, NBLK);

    // Copy the result from device memory to host memory.
    //...
    err = hipMemcpy(this->data.data(), d_C, sizeBytes, hipMemcpyDeviceToHost);
    ERR_CHK(err, "Failed to copy data back from GPU to CPU");

    // Free device memory.
    err = hipFree(d_A);
    ERR_CHK (err, "Failed to free CUDA matrix A");
    //...
    err = hipFree(d_B);
    ERR_CHK(err, "Failed to free CUDA matrix B");
    err = hipFree(d_C);
    ERR_CHK(err, "Failed to free CUDA matrix C");
}
